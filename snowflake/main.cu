#include "hip/hip_runtime.h"
#include "function.h"
int main(int argc,char* argv[]){
   highprecision *phi,*phi_lap,*tempr,*tempr_lap,*phidx,*phidy,*epsilon,*epsilon_deri;
   CHECK_ERROR(hipMallocManaged((void**)&phi,sizeof(highprecision)*dimX*dimY));
   CHECK_ERROR(hipMallocManaged((void**)&phi_lap,sizeof(highprecision)*dimX*dimY));
   CHECK_ERROR(hipMallocManaged((void**)&tempr,sizeof(highprecision)*dimX*dimY));
   CHECK_ERROR(hipMallocManaged((void**)&tempr_lap,sizeof(highprecision)*dimX*dimY));
   CHECK_ERROR(hipMallocManaged((void**)&phidx,sizeof(highprecision)*dimX*dimY));
   CHECK_ERROR(hipMallocManaged((void**)&phidy,sizeof(highprecision)*dimX*dimY));
   CHECK_ERROR(hipMallocManaged((void**)&epsilon,sizeof(highprecision)*dimX*dimY));
   CHECK_ERROR(hipMallocManaged((void**)&epsilon_deri,sizeof(highprecision)*dimX*dimY));
   for(int y=0;y<dimY;y++){
      for(int x=0;x<dimX;x++){
         if(pow(y-(dimY/2+8),2)+pow(x-(dimX/2+8),2)<seed){
            phi[y*dimX+x]=1.0;
         }
      }
    }
    dim3 blocks_pure(unitx,unity);
    dim3 grids_pure(1,1,unitdimX*unitdimY);
    for(int i=0;i<timesteps;i++){
        kernel1_pure<<<grids_pure,blocks_pure>>>(phi,phi_lap,tempr,tempr_lap,phidx,phidy,epsilon,epsilon_deri);
        hipDeviceSynchronize();
        kernel2_pure<<<grids_pure,blocks_pure>>>(phi,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap);
        hipDeviceSynchronize();


    }
    CHECK_ERROR(hipFree(phi));CHECK_ERROR(hipFree(phi_lap));CHECK_ERROR(hipFree(tempr));
    CHECK_ERROR(hipFree(tempr_lap));CHECK_ERROR(hipFree(phidx));CHECK_ERROR(hipFree(phidy));
    CHECK_ERROR(hipFree(epsilon));CHECK_ERROR(hipFree(epsilon_deri));
    return 0;
}