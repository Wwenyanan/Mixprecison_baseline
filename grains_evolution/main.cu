#include "hip/hip_runtime.h"
#include "function.h"
int main(int argc,char* argv[]){
    //空间申请
    highprecision *eta1,*eta2,*eta1_lap,*eta2_lap,*dfdeta1,*dfdeta2,*eta1_out,*eta2_out;
    CHECK_ERROR(hipMallocManaged((void**)&eta1,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_out,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_out,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta1,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta2,sizeof(highprecision)*dimX*dimY));
    // 数据准备
    for(int y=1;y<=dimY;y++){
        for(int x=1;x<=dimX;x++){
            float dis1=sqrt(pow(x-Rx,2)+pow(y-Ry,2));
            if(dis1<=R){
                eta1[(y-1)*dimX+x-1]=0;
                eta2[(y-1)*dimX+x-1]=1;
            }
            else{
                eta1[(y-1)*dimX+x-1]=1;
                eta2[(y-1)*dimX+x-1]=0;
            }

        }
    }


    // 线程数量设置
    dim3 blocks_pure(unitx,unity);
    dim3 grids_pure(1,1,unitdimX*unitdimY);
    for(int i=0;i<timesteps;i++){
        kernel1_pure<<<grids_pure,blocks_pure>>>(eta1,eta2,eta1_lap,dfdeta1,eta1_out);
        hipDeviceSynchronize();
        kernel1_pure<<<grids_pure,blocks_pure>>>(eta2,eta1,eta2_lap,dfdeta2,eta2_out);
        hipDeviceSynchronize();
        swap(eta1,eta1_out);
        swap(eta2,eta2_out);
    }

    // 释放空间
    CHECK_ERROR(hipFree(eta1));
    CHECK_ERROR(hipFree(eta2));
    CHECK_ERROR(hipFree(eta1_out));
    CHECK_ERROR(hipFree(eta2_out));
    CHECK_ERROR(hipFree(eta1_lap));
    CHECK_ERROR(hipFree(eta2_lap));
    CHECK_ERROR(hipFree(dfdeta1));
    CHECK_ERROR(hipFree(dfdeta2));
    return 0;

}