#include "hip/hip_runtime.h"
#include "function.h"
int main(int argc,char* argv[]){
    //空间申请
    highprecision *con,*eta1,*eta2,*eta1_lap,*eta2_lap,*con_lap,*dummy,*dummy_lap,*dfdcon,*dfdeta1,*dfdeta2,*eta1_out,*eta2_out;
    CHECK_ERROR(hipMallocManaged((void**)&con,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_out,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_out,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dummy,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&con_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dummy_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdcon,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta1,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta2,sizeof(highprecision)*dimX*dimY));


    // 数据准备
    for(int y=1;y<=dimY;y++){
        for(int x=1;x<=dimX;x++){
            float dis1=sqrt(pow(x-Rx1,2)+pow(y-Ry1,2));
            float dis2=sqrt(pow(x-Rx1,2)+pow(y-Ry2,2));
            if(dis1<=R1){
                con[(y-1)*dimX+x-1]=1;
                eta1[(y-1)*dimX+x-1]=1;
            }
            if(dis2<=R2){
                con[(y-1)*dimX+x-1]=1;
                eta2[(y-1)*dimX+x-1]=1;
                eta1[(y-1)*dimX+x-1]=0.0;
            }
        }
    }
    // 线程数量设置
    dim3 blocks_pure(unitx,unity);
    dim3 grids_pure(1,1,unitdimX*unitdimY);

    for(int i=0;i<timesteps;i++){
        con1_pure<<<grids_pure,blocks_pure>>>(con,con_lap,eta1,eta2,dfdcon,dummy,i);
        hipDeviceSynchronize();
        con2_pure<<<grids_pure,blocks_pure>>>(dummy,dummy_lap,con,eta1,eta2,i);
        hipDeviceSynchronize();
        phi1_pure<<<grids_pure,blocks_pure>>>(eta1,eta1_out,eta2,eta1_lap,dfdeta1,con,i);
        hipDeviceSynchronize();
        phi2_pure<<<grids_pure,blocks_pure>>>(eta2,eta2_out,eta1,eta2_lap,dfdeta2,con,i);
        hipDeviceSynchronize();
        swap(eta1,eta1_out);
        swap(eta2,eta2_out);
    }

    // 释放空间
    CHECK_ERROR(hipFree(con));
    CHECK_ERROR(hipFree(eta1));
    CHECK_ERROR(hipFree(eta2));
    CHECK_ERROR(hipFree(eta1_out));
    CHECK_ERROR(hipFree(eta2_out));
    CHECK_ERROR(hipFree(dummy));
    CHECK_ERROR(hipFree(con_lap));
    CHECK_ERROR(hipFree(eta1_lap));
    CHECK_ERROR(hipFree(eta2_lap));
    CHECK_ERROR(hipFree(dummy_lap));
    CHECK_ERROR(hipFree(dfdcon));
    CHECK_ERROR(hipFree(dfdeta1));
    CHECK_ERROR(hipFree(dfdeta2));
    return 0;
}